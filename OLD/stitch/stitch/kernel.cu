#include "hip/hip_runtime.h"
#include <opencv2/core/cuda_devptrs.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/gpu/gpu.hpp>
#include <opencv2/gpu/stream_accessor.hpp>
#include <opencv2/opencv.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/gpu/devmem2d.hpp>
#include <opencv2/gpu/gpumat.hpp>


#define MIDDLEWIDTH 100
#define LEFT 1256
#define RIGHT 723
#define LARGEST1 1340
#define LARGEST2 600
using namespace cv;
using namespace cv::gpu;
using namespace std;
__global__ void stitch_kernel(const PtrStepSz<uchar3> src1, const PtrStepSz<uchar3> src2, PtrStep<uchar3> dst, int limit)
{
	int abs_x = threadIdx.x + blockDim.x * blockIdx.x;
	int abs_y = threadIdx.y + blockDim.y * blockIdx.y;
	if (abs_y < 0 || abs_y >= limit || abs_x < 0 || abs_x >= MIDDLEWIDTH) {
		return;

	}
	int pointOnLeft = abs_x + LEFT;
	int pointOnRight = RIGHT - MIDDLEWIDTH + abs_x;
	uchar3 value1 = src1(abs_y, pointOnLeft);
	uchar3 value2 = src2(abs_y, pointOnRight);
	unsigned char newRed = (((LEFT + MIDDLEWIDTH) - pointOnLeft) / (MIDDLEWIDTH*1.0)) * value1.x + ((pointOnLeft - LEFT) / (MIDDLEWIDTH*1.0)) * value2.x;
	unsigned char newGreen = (((LEFT + MIDDLEWIDTH) - pointOnLeft) / (MIDDLEWIDTH*1.0)) * value1.y + ((pointOnLeft - LEFT) / (MIDDLEWIDTH*1.0)) * value2.y;
	unsigned char newBlue = (((LEFT + MIDDLEWIDTH) - pointOnLeft) / (MIDDLEWIDTH*1.0)) * value1.z + ((pointOnLeft - LEFT) / (MIDDLEWIDTH*1.0)) * value2.z;
	dst(abs_y, pointOnLeft) = make_uchar3(newRed, newGreen, newBlue);

}

void stitch_caller(const PtrStepSz<uchar3>& src1, const PtrStepSz<uchar3>& src2, PtrStep<uchar3> dst, int limit, hipStream_t stream)
{
	int blockWidth = 32;
	int blockHeight = 8;
	dim3 blockSize(blockWidth, blockHeight);
	int blocksY = src1.rows / blockHeight;
	int blocksX = src1.cols / blockWidth;
	const dim3 gridSize(blocksX, blocksY);
	stitch_kernel << <gridSize, blockSize, 0, stream >> >(src1, src2, dst, limit);
	if (stream == 0)
	{
		hipDeviceSynchronize();
	}
	
}
void stitch(const GpuMat& src1, const GpuMat& src2, GpuMat& dst, int limit, Stream& stream = Stream::Null())
{
	CV_Assert(src1.type() == CV_8UC3);
	CV_Assert(src2.type() == CV_8UC3);
	hipStream_t s = StreamAccessor::getStream(stream);
	stitch_caller(src1, src2, dst, limit, s);
}
int main()
{	
	Mat imageRight = imread("D:\\Documents\\Images\\resulti_phi.jpg");
	Mat imageLeft = imread("D:\\Documents\\Images\\resultii_phi.jpg");
	int width = LEFT + MIDDLEWIDTH + imageRight.cols - RIGHT;
	Mat combine(max(imageRight.rows, imageLeft.rows), width, CV_8UC3, Scalar(0, 0, 0));
	GpuMat gpuMatRight, gpuMatLeft, output;
	gpuMatRight.upload(imageRight);
	gpuMatLeft.upload(imageLeft);
	output.upload(combine);
	GpuMat left_roi(output, Rect(0, 0, LEFT, gpuMatLeft.size().height));
	Size middleSize(MIDDLEWIDTH, gpuMatLeft.size().height);
	GpuMat middleZone(middleSize, CV_8UC3, Scalar(0, 0, 0));
	GpuMat croppedImage1;
	Rect roi1(0, 0, LEFT, imageLeft.rows);
	croppedImage1 = gpuMatLeft(roi1);
	croppedImage1.copyTo(left_roi);
	GpuMat right_roi(output, Rect(LEFT + MIDDLEWIDTH, 0, imageRight.cols - RIGHT, imageRight.rows));
	GpuMat croppedImage2;
	Rect roi2(RIGHT, 0, imageRight.cols - RIGHT, imageRight.rows);
	croppedImage2 = gpuMatRight(roi2);
	croppedImage2.copyTo(right_roi);
	stitch(gpuMatLeft, gpuMatRight, output, gpuMatLeft.rows);
	output.download(combine);
	imwrite("D:\\Documents\\Images\\temp.jpg", combine);

	return 0;
}