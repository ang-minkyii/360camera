#include "hip/hip_runtime.h"
//#include <iostream>
#include <numeric>
#include <stdlib.h>
//#include "utils.h"
//#include <string>
#include <stdio.h>

//#include <opencv2/highgui/highgui.hpp>
//#include <opencv2/imgproc/imgproc.hpp>
//#include <opencv2/opencv.hpp>

//using namespace cv;
#include "functions.hpp"
#define PI 3.1415926
#define RADIUS 770
#define centreX 776
#define centreY 540
double cameraFieldAngle = PI;
double latitudeOffset = (PI - cameraFieldAngle)/2;
double longitudeOffset = (PI - cameraFieldAngle)/2;

__global__
void correction(const uchar4* const inputImage,
				uchar4* const rectImage,
				int numRows, int numCols,double dx, double dy,double latitudeOffset,double longitudeOffset)
{
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	double x,y,z,r;
	double latitude,longitude;
	double u,v;
	double u_latitude,v_latitude;
	double theta_sphere;
	double phi_shpere;
	double theta;
	double x_cart;
	double y_cart;
	double p;
	latitude = latitudeOffset + j * dy;
	longitude = latitudeOffset + i * dx;

	x = -sin(latitude) * cos(longitude);
	y = cos(latitude);
	z = sin(latitude) * sin(longitude);

	theta_sphere = acos(z);
	phi_shpere = atan2(y,x);
	//phi_shpere = phi_shpere * PI/180;
	p = sin(theta_sphere);
	theta = phi_shpere;

	x_cart = p*cos(theta);
	y_cart = p*sin(theta);
	double R = RADIUS;
	u = x_cart *R + centreX;
	v = -y_cart *R + centreY;





	if(j < numCols && i < numRows  && v < numRows && u < numCols){
		int newIndex = numRows * v + u;
		int index = numRows * j + i;
		uchar4 value = inputImage[index];
		rectImage[newIndex] = value;
	}


}
/*
 *
 * Original CPU Code of correction
 *
 * Mat correct(Mat imageInput){
    double camerFieldAngle = PI;

    double rateOfWindow = 1;

    //int width = imageInput.size().width*rateOfWindow;
    int width = imageInput.size().width * rateOfWindow;
    int height = width;
    Size imgSize(width,height);
    Mat retImg(imgSize,CV_8UC3,Scalar(0,0,0));
    double dx = camerFieldAngle/imgSize.width;
    double dy = dx;

    double latitude;
    double longitude;

    double x,y,z,r;
    double Theta_sphere;
    double Phi_sphere;
    double p;
    double theta;
    double x_cart;
    double y_cart;
    int u,v;
    int u_latitude, v_latitude;

    double longitude_offset, latitude_offset;

    longitude_offset = (PI-camerFieldAngle)/2;
    latitude_offset = (PI - camerFieldAngle)/2;

    Mat_<Vec3b> _retImg = retImg;
    Mat_<Vec3b> _imageInput = imageInput;

    for(int j = 0; j < imgSize.height;j++){
        latitude = latitude_offset + j * dy;
        for(int i = 0; i < imgSize.width;i++){
            longitude = latitude_offset + i * dx;
            x = -sin(latitude) * cos(longitude);
            y = cos(latitude);
            z = sin(latitude) * sin(longitude);

            Theta_sphere = acos(z);
            Phi_sphere = cvFastArctan(y, x);
            Phi_sphere = Phi_sphere * PI /180;
            p = sin(Theta_sphere);
            theta = Phi_sphere;

            x_cart = p * cos(theta);
            y_cart = p * sin(theta);
            double R = RADIUS;
            u = x_cart * R + centreX;
            v = -y_cart * R + centreY;
            _retImg.at<Vec3b>(j,i) = _imageInput.at<Vec3b>(v,u);
        }
    }
    return retImg;
}
 * */
void OverallCorrection(const uchar4* const h_inputImage,
		uchar4* const d_inputImage,uchar4* const d_rectImage,
		size_t numRows, size_t numCols){
	int blockWidth = 32;
	const dim3 blockSize(blockWidth, blockWidth, 1);
	int blocksX = numRows/blockWidth + 1;
	int blocksY = numCols/blockWidth + 1;
	const dim3 gridSize(blocksX, blocksY, 1);
	double dx = cameraFieldAngle/numCols;
	double dy = cameraFieldAngle/numRows;
	correction<<<gridSize,blockSize>>>(d_inputImage,d_rectImage,numRows, numCols,dx,dy,latitudeOffset, longitudeOffset);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());
}
int main(void)
{
	//Mat img(512,512,CV_8UC3,Scalar(0));
	//putText(img,"Hello",Point(10,img.rows/2),FONT_HERSHEY_DUPLEX,1.0,CV_RGB(118,185,0),2);
	//imshow("hello",img);
	//Mat img2;
	//img2 = imread("my_photo-1.jpg");
	//imshow("image2",img2);
	//imwrite("result.jpg",img);
	//waitKey();
	Mat img1 = imread("capture.jpg");
	imwrite("cpimg1.jpg",img1);
	uchar4 *h_inputImage, *d_inputImage;
	uchar4 *h_rectImage, *d_rectImage;

	std::string input_file;
	std::string output_file;
	input_file = "capture.jpg";
	output_file = "Output.jpg";

	preProcess(&h_inputImage, &h_rectImage, &d_inputImage, &d_rectImage,input_file);
	OverallCorrection(h_inputImage,d_inputImage,d_rectImage,height(),width());
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	size_t numPixels = height() * width();
	checkCudaErrors(hipMemcpy(h_rectImage,d_rectImage,sizeof(uchar4)* numPixels,hipMemcpyDeviceToHost));

	postProcess(output_file,h_rectImage);
	cleanUp();
	return 0;
}

