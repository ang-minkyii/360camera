#include "hip/hip_runtime.h"

#include <iostream>  
#include <string>  
#include <vector>  
#include <algorithm>  
#include <numeric>  
#include <opencv2/core/core.hpp>  
#include <opencv2/imgproc/imgproc.hpp>
//#include <opencv2/core/opengl_interop.hpp> 
#include <opencv2/core/cuda_devptrs.hpp>
#include <opencv2/gpu/gpu.hpp>  
#include <opencv2/highgui/highgui.hpp>  
#include <opencv2/contrib/contrib.hpp> 
#include <opencv2/gpu/stream_accessor.hpp>
#include <opencv2/gpu/gpumat.hpp>
#include <time.h>
#include "parameters.h"
#define WIDTH2 1920
#define HEIGHT2 1920
#define FINALWIDTH 3874
#define FINALHEIGHT 1920
#define LEFT1 1261
#define LEFT2 1909
#define LEFT3 2564
#define RIGHT 660
#define MIDDLEWIDTH 50

using namespace cv;
using namespace cv::gpu;
#include<iostream>  
using namespace std;
__global__ void stitch_kernel(const PtrStepSz<uchar3> src1, const PtrStepSz<uchar3> src2, PtrStep<uchar3> dst, int limit, int left)
{
	int abs_x = threadIdx.x + blockDim.x * blockIdx.x;
	int abs_y = threadIdx.y + blockDim.y * blockIdx.y;
	if (abs_y < 0 || abs_y >= limit || abs_x < 0 || abs_x >= MIDDLEWIDTH) {
		return;

	}
	int pointOnLeft = abs_x + left;
	int pointOnRight = RIGHT - MIDDLEWIDTH + abs_x;
	uchar3 value1 = src1(abs_y, pointOnLeft);
	uchar3 value2 = src2(abs_y, pointOnRight);
	unsigned char newRed = (((left + MIDDLEWIDTH) - pointOnLeft) / (MIDDLEWIDTH*1.0)) * value1.x + ((pointOnLeft - left) / (MIDDLEWIDTH*1.0)) * value2.x;
	unsigned char newGreen = (((left + MIDDLEWIDTH) - pointOnLeft) / (MIDDLEWIDTH*1.0)) * value1.y + ((pointOnLeft - left) / (MIDDLEWIDTH*1.0)) * value2.y;
	unsigned char newBlue = (((left + MIDDLEWIDTH) - pointOnLeft) / (MIDDLEWIDTH*1.0)) * value1.z + ((pointOnLeft - left) / (MIDDLEWIDTH*1.0)) * value2.z;
	dst(abs_y, pointOnLeft) = make_uchar3(newRed, newGreen, newBlue);

}

void stitch_caller(const PtrStepSz<uchar3>& src1, const PtrStepSz<uchar3>& src2, PtrStep<uchar3> dst, int limit, int left, hipStream_t stream)
{
	int blockWidth = 32;
	int blockHeight = 8;
	dim3 blockSize(blockWidth, blockHeight);
	int blocksY = src1.rows / blockHeight;
	int blocksX = src1.cols / blockWidth;
	const dim3 gridSize(blocksX, blocksY);
	stitch_kernel << <gridSize, blockSize, 0, stream >> >(src1, src2, dst, limit, left);
	if (stream == 0)
	{
		hipDeviceSynchronize();
	}

}
void stitch(const GpuMat& src1, const GpuMat& src2, GpuMat& dst, int limit, int left, Stream& stream = Stream::Null())
{
	CV_Assert(src1.type() == CV_8UC3);
	CV_Assert(src2.type() == CV_8UC3);
	hipStream_t s = StreamAccessor::getStream(stream);
	stitch_caller(src1, src2, dst, limit, left, s);
}
int main(int argc, char* argv[])
{

	const string fname = "D:\\Documents\\Movies\\out1.avi";
	const string fname2 = "D:\\Documents\\Movies\\out2.avi";
	const string fname3 = "D:\\Documents\\Movies\\out3.avi";
	const string fname4 = "D:\\Documents\\Movies\\out4.avi";
	std::string fname5 = "D:\\Documents\\Movies\\Project\\attempt3\\1.avi";
	const string fname6 = "D:\\Documents\\Movies\\Project\\attempt3\\2.avi";
	const string fname7 = "D:\\Documents\\Movies\\Project\\attempt3\\3.avi";
	const string fname8 = "D:\\Documents\\Movies\\Project\\attempt3\\4.avi";
	time_t finish, start;
	//string mm = cv::getBuildInformation();
	//cout << mm << endl;
	//cv::namedWindow("CPU", cv::WINDOW_NORMAL);
	//cv::namedWindow("GPU", cv::WINDOW_OPENGL);
	//gpu::setGlDevice();
	Mat frame,frame2,frame3,frame4;
	Mat framecorrected(HEIGHT2, WIDTH2, CV_8UC3, Scalar(0, 0, 0));
	Mat framecorrected2(HEIGHT2, WIDTH2, CV_8UC3, Scalar(0, 0, 0));
	Mat framecorrected3(HEIGHT2, WIDTH2, CV_8UC3, Scalar(0, 0, 0));
	Mat framecorrected4(HEIGHT2, WIDTH2, CV_8UC3, Scalar(0, 0, 0));
	GpuMat frame_d, frame2_d, frame3_d, frame4_d;
	GpuMat framecorrected_d, framecorrected2_d, framecorrected3_d, framecorrected4_d;
	Mat result(1566, 640, CV_8UC3, Scalar(0, 0, 0));
	cv::VideoCapture reader(fname5);
	cv::VideoCapture reader2(fname6);
	cv::VideoCapture reader3(fname7);
	cv::VideoCapture reader4(fname8);
	CvSize frameSize = cvSize(int(reader.get(CV_CAP_PROP_FRAME_WIDTH)), int(reader.get(CV_CAP_PROP_FRAME_HEIGHT)));
	printf("height:%d\n", frameSize.height);
	printf("width:%d\n", frameSize.width);
	double count = reader.get(CV_CAP_PROP_FRAME_COUNT);
	printf("number of frames = %d\n", count);
	int n = 0;
	//reader.set(CV_CAP_PROP_POS_FRAMES, count - 1);
//	cv::gpu::GpuMat d_frame;
//	cv::gpu::VideoReader_GPU d_reader(fname);
	//d_reader.dumpFormat(std::cout);

	cv::TickMeter tm;
	std::vector<double> cpu_times;
//	std::vector<double> gpu_times;
	
	
	cv::Mat x_map2(HEIGHT2, WIDTH2, CV_32FC1, &_x_map_1920);
	cv::Mat y_map2(HEIGHT2, WIDTH2, CV_32FC1, &_y_map_1920);
	GpuMat x_map2_d, y_map2_d;
	x_map2_d.upload(x_map2);
	y_map2_d.upload(y_map2);
	for (;;)
	{
		tm.reset(); tm.start();
		if (!reader.read(frame) || !reader2.read(frame2) || !reader3.read(frame3) || !reader4.read(frame4))
			break;
		
		/*if (n == 0)
		{
			cv::imwrite("D:\\Documents\\Images\\attempt3_5.jpg",frame);
		}
		n++;*/
		tm.stop();
		cpu_times.push_back(tm.getTimeMilli());

	//	tm.reset(); tm.start();
		//if (!d_reader.read(d_frame))
			//break;
		//tm.stop();
		//gpu_times.push_back(tm.getTimeMilli());
		time(&start);
		frame_d.upload(frame);
		frame2_d.upload(frame2);
		frame3_d.upload(frame3);
		frame4_d.upload(frame4);
		remap(frame_d, framecorrected_d, x_map2_d, y_map2_d, INTER_LINEAR);
		remap(frame2_d, framecorrected2_d, x_map2_d, y_map2_d, INTER_LINEAR);
		remap(frame3_d, framecorrected3_d, x_map2_d, y_map2_d, INTER_LINEAR);
		remap(frame4_d, framecorrected4_d, x_map2_d, y_map2_d, INTER_LINEAR);
	//	int width = LEFT1 + MIDDLEWIDTH + framecorrected.cols - RIGHT;
		Mat result(FINALHEIGHT, FINALWIDTH, CV_8UC3, Scalar(0, 0, 0));
		//GpuMat combine(1920, width, CV_8UC3, Scalar(0, 0, 0));
		GpuMat result_d(FINALHEIGHT, FINALWIDTH, CV_8UC3, Scalar(0, 0, 0));
		GpuMat left_roi_d(result_d, Rect(0, 0, LEFT1, framecorrected_d.size().height));
		Size middleSize(MIDDLEWIDTH, framecorrected_d.size().height);
		GpuMat middle_zone_d(middleSize, CV_8UC3, Scalar(0, 0, 0));
		GpuMat croppedImage1;
		Rect Roi1(0, 0, LEFT1, framecorrected_d.size().height);
		croppedImage1 = framecorrected_d(Roi1);
		croppedImage1.copyTo(left_roi_d);
		GpuMat second_roi_d(result_d,Rect(LEFT1+MIDDLEWIDTH,0,framecorrected2_d.cols-RIGHT,framecorrected2_d.rows));
		GpuMat croppedImage2;
		Rect Roi2(RIGHT, 0, framecorrected2_d.cols - RIGHT, framecorrected2_d.rows);
		croppedImage2 = framecorrected2_d(Roi2);
		croppedImage2.copyTo(second_roi_d);
		stitch(framecorrected_d, framecorrected2_d, result_d, framecorrected_d.rows, LEFT1);

		GpuMat third_roi_d(result_d, Rect(LEFT2 + MIDDLEWIDTH, 0, framecorrected3_d.cols - RIGHT, framecorrected3_d.rows));
		GpuMat croppedImage3;
		croppedImage3 = framecorrected3_d(Roi2);
		croppedImage3.copyTo(third_roi_d);
		stitch(result_d, framecorrected3_d, result_d, result_d.rows, LEFT2);

		GpuMat fourth_roi_d(result_d, Rect(LEFT3 + MIDDLEWIDTH, 0, framecorrected4_d.cols - RIGHT, framecorrected4_d.rows));
		GpuMat croppedImage4 = framecorrected4_d(Roi2);
		croppedImage4.copyTo(fourth_roi_d);
		stitch(result_d, framecorrected4_d, result_d, result_d.rows, LEFT3);

		result_d.download(result);
		//framecorrected_d.download(framecorrected);
		//framecorrected2_d.download(framecorrected2);
		//framecorrected3_d.download(framecorrected3);
		//framecorrected4_d.download(framecorrected4);
		time(&finish);
		/*remap(frame, framecorrected, x_map2, y_map2, INTER_LINEAR);
		remap(frame2, framecorrected2, x_map2, y_map2, INTER_LINEAR);
		remap(frame3, framecorrected3, x_map2, y_map2, INTER_LINEAR);
		remap(frame4, framecorrected4, x_map2, y_map2, INTER_LINEAR);*/
		//namedWindow("CPU", WINDOW_NORMAL);
		//namedWindow("CPU2", WINDOW_NORMAL);
		//namedWindow("CPU3", WINDOW_NORMAL);
		//namedWindow("CPU4", WINDOW_NORMAL);
		//cv::imshow("CPU", framecorrected);
		//imshow("CPU2", framecorrected2);
		//imshow("CPU3", framecorrected3);
		//imshow("CPU4", framecorrected4);
		namedWindow("result", WINDOW_NORMAL);
		imshow("result", result);
		/**Mat left_roi(res, Rect(0, 0, LEFT1, src.size().height));
		Size middleSize(MIDDLEWIDTH, src.size().height);//added
		Mat middleZone(middleSize, CV_8UC3, Scalar(0, 0, 0));//added
		Mat croppedImage1;

		// Rect Roi1(0,0,LEFT,image2.rows);
		Rect Roi1(0, 0, LEFT1, src.rows);
		// croppedImage1 = image2(Roi1);
		croppedImage1 = src(Roi1);
		croppedImage1.copyTo(left_roi);
		//printf("largest1 = %d, %d,%d\n",(int)largest1,(int)(image1.cols-largest2),image1.rows);
		// Mat right_roi(combine,Rect(LEFT+MIDDLEWIDTH,0,image1.cols - RIGHT,image1.rows));
		Mat second_roi(res, Rect(LEFT1 + MIDDLEWIDTH, 0, src2.cols - RIGHT, src2.rows));
		//Mat right_roi(combine,Rect(largest1,0,image1.cols-largest2,image1.rows));
		Mat croppedImage2;
		//Rect Roi2(RIGHT,0,image1.cols-RIGHT,image1.rows);
		Rect Roi2(RIGHT, 0, src2.cols - RIGHT, src2.rows);
		//croppedImage2 = image1(Roi2);
		croppedImage2 = src2(Roi2);
		// croppedImage2.copyTo(right_roi);
		croppedImage2.copyTo(second_roi);
		//imshow("combine",combine);
		//imwrite("combined.jpg",combine);

		//Mat_<Vec3b> _orgImg1 = image2;
		// Mat_<Vec3b> _orgImg2 = image1;
		Mat_<Vec3b> _orgImg1 = src;
		Mat_<Vec3b> _orgImg2 = src2;
		//Mat_<Vec3b> _retImg = combine;
		Mat_<Vec3b> _retImg = res;
		//vector<Mat> channels;
		//split(combine,channels);
		for (int j = 0; j < res.rows; j++) {
			for (int i = 0; i < MIDDLEWIDTH; i++) {
				//int pointOnLeft = i + LEFT;
				int pointOnLeft = i + LEFT1;
				int pointOnRight = RIGHT - MIDDLEWIDTH + i;
				int RedOne = _orgImg1.at<Vec3b>(j, pointOnLeft)[0];
				int GreenOne = _orgImg1.at<Vec3b>(j, pointOnLeft)[1];
				int YellowOne = _orgImg1.at<Vec3b>(j, pointOnLeft)[2];
				int RedTwo = _orgImg2.at<Vec3b>(j, pointOnRight)[0];
				int GreenTwo = _orgImg2.at<Vec3b>(j, pointOnRight)[1];
				int YellowTwo = _orgImg2.at<Vec3b>(j, pointOnRight)[2];
				double newRed = (((LEFT1 + MIDDLEWIDTH) - pointOnLeft) / (MIDDLEWIDTH*1.0)) * RedOne + ((pointOnLeft - LEFT1) / (MIDDLEWIDTH*1.0)) * RedTwo;
				double newGreen = ((LEFT1 + MIDDLEWIDTH - pointOnLeft) / (MIDDLEWIDTH*1.0)) * GreenOne + ((pointOnLeft - LEFT1) / (MIDDLEWIDTH*1.0)) * GreenTwo;
				double newYellow = ((LEFT1 + MIDDLEWIDTH - pointOnLeft) / (MIDDLEWIDTH*1.0)) * YellowOne + ((pointOnLeft - LEFT1) / (MIDDLEWIDTH*1.0)) * YellowTwo;
			//	double newRed = (((LEFT+MIDDLEWIDTH) - pointOnLeft)/(MIDDLEWIDTH*1.0)) * RedOne + ((pointOnLeft- LEFT)/(MIDDLEWIDTH*1.0)) * RedTwo;
			//	double newGreen = ((LEFT+MIDDLEWIDTH-pointOnLeft)/(MIDDLEWIDTH*1.0)) * GreenOne + ((pointOnLeft- LEFT1)/(MIDDLEWIDTH*1.0)) * GreenTwo;
			//	double newYellow = ((LEFT+MIDDLEWIDTH-pointOnLeft)/(MIDDLEWIDTH*1.0)) * YellowOne + ((pointOnLeft- LEFT)/(MIDDLEWIDTH*1.0)) * YellowTwo;
				_retImg.at<Vec3b>(j, pointOnLeft)[0] = (int)newRed;
				_retImg.at<Vec3b>(j, pointOnLeft)[1] = (int)newGreen;
				_retImg.at<Vec3b>(j, pointOnLeft)[2] = (int)newYellow;


			}
		}
		Mat third_roi(res, Rect(LEFT2 + MIDDLEWIDTH, 0, src3.cols - RIGHT, src3.rows));
		Mat croppedImage3 = src3(Roi2);
		croppedImage3.copyTo(third_roi);
		_orgImg1 = res;
		_orgImg2 = src3;
		_retImg = res;
		for (int j = 0; j < res.rows; j++) {
			for (int i = 0; i < MIDDLEWIDTH; i++) {
				//int pointOnLeft = i + LEFT;
				int pointOnLeft = i + LEFT2;
				int pointOnRight = RIGHT - MIDDLEWIDTH + i;
				int RedOne = _orgImg1.at<Vec3b>(j, pointOnLeft)[0];
				int GreenOne = _orgImg1.at<Vec3b>(j, pointOnLeft)[1];
				int YellowOne = _orgImg1.at<Vec3b>(j, pointOnLeft)[2];
				int RedTwo = _orgImg2.at<Vec3b>(j, pointOnRight)[0];
				int GreenTwo = _orgImg2.at<Vec3b>(j, pointOnRight)[1];
				int YellowTwo = _orgImg2.at<Vec3b>(j, pointOnRight)[2];
				double newRed = (((LEFT2 + MIDDLEWIDTH) - pointOnLeft) / (MIDDLEWIDTH*1.0)) * RedOne + ((pointOnLeft - LEFT2) / (MIDDLEWIDTH*1.0)) * RedTwo;
				double newGreen = ((LEFT2 + MIDDLEWIDTH - pointOnLeft) / (MIDDLEWIDTH*1.0)) * GreenOne + ((pointOnLeft - LEFT2) / (MIDDLEWIDTH*1.0)) * GreenTwo;
				double newYellow = ((LEFT2 + MIDDLEWIDTH - pointOnLeft) / (MIDDLEWIDTH*1.0)) * YellowOne + ((pointOnLeft - LEFT2) / (MIDDLEWIDTH*1.0)) * YellowTwo;
				_retImg.at<Vec3b>(j, pointOnLeft)[0] = (int)newRed;
				_retImg.at<Vec3b>(j, pointOnLeft)[1] = (int)newGreen;
				_retImg.at<Vec3b>(j, pointOnLeft)[2] = (int)newYellow;


			}
		}
		Mat fourth_roi(res, Rect(LEFT3 + MIDDLEWIDTH, 0, src4.cols - RIGHT, src4.rows));
		Mat croppedImage4 = src4(Roi2);
		croppedImage4.copyTo(fourth_roi);
		_orgImg1 = res;
		_orgImg2 = src4;
		_retImg = res;
		for (int j = 0; j < res.rows; j++) {
			for (int i = 0; i < MIDDLEWIDTH; i++) {
				//int pointOnLeft = i + LEFT;
				int pointOnLeft = i + LEFT3;
				int pointOnRight = RIGHT - MIDDLEWIDTH + i;
				int RedOne = _orgImg1.at<Vec3b>(j, pointOnLeft)[0];
				int GreenOne = _orgImg1.at<Vec3b>(j, pointOnLeft)[1];
				int YellowOne = _orgImg1.at<Vec3b>(j, pointOnLeft)[2];
				int RedTwo = _orgImg2.at<Vec3b>(j, pointOnRight)[0];
				int GreenTwo = _orgImg2.at<Vec3b>(j, pointOnRight)[1];
				int YellowTwo = _orgImg2.at<Vec3b>(j, pointOnRight)[2];
				double newRed = (((LEFT3 + MIDDLEWIDTH) - pointOnLeft) / (MIDDLEWIDTH*1.0)) * RedOne + ((pointOnLeft - LEFT3) / (MIDDLEWIDTH*1.0)) * RedTwo;
				double newGreen = ((LEFT3 + MIDDLEWIDTH - pointOnLeft) / (MIDDLEWIDTH*1.0)) * GreenOne + ((pointOnLeft - LEFT3) / (MIDDLEWIDTH*1.0)) * GreenTwo;
				double newYellow = ((LEFT3 + MIDDLEWIDTH - pointOnLeft) / (MIDDLEWIDTH*1.0)) * YellowOne + ((pointOnLeft - LEFT3) / (MIDDLEWIDTH*1.0)) * YellowTwo;
				_retImg.at<Vec3b>(j, pointOnLeft)[0] = (int)newRed;
				_retImg.at<Vec3b>(j, pointOnLeft)[1] = (int)newGreen;
				_retImg.at<Vec3b>(j, pointOnLeft)[2] = (int)newYellow;


			}
		}*/
		//cv::imshow("GPU", d_frame);

		if (cv::waitKey(3) > 0)
			break;
	}

	if (!cpu_times.empty() )//&& !gpu_times.empty())
	{
		std::cout << std::endl << "Results:" << std::endl;

		std::sort(cpu_times.begin(), cpu_times.end());
//		std::sort(gpu_times.begin(), gpu_times.end());

		double cpu_avg = std::accumulate(cpu_times.begin(), cpu_times.end(), 0.0) / cpu_times.size();
	//	double gpu_avg = std::accumulate(gpu_times.begin(), gpu_times.end(), 0.0) / gpu_times.size();

		std::cout << "CPU : Avg : " << cpu_avg << " ms FPS : " << 1000.0 / cpu_avg << std::endl;
	//std::cout << "GPU : Avg : " << gpu_avg << " ms FPS : " << 1000.0 / gpu_avg << std::endl;
	}
	printf("difference in time = %d\n", difftime(finish,start));
	cvDestroyAllWindows();
	system("pause");
	return 0;
}