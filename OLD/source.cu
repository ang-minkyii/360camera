#include "hip/hip_runtime.h"
/*
 * source.cu
 *
 *  Created on: 9 Feb 2017
 *      Author: andy
 */

#include "Capture.hpp"
#include <cstdio>
#include <ctime>

/*
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <iostream>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/gpu/gpu.hpp>

#include <getopt.h>

#include <fcntl.h>
#include <unistd.h>
#include <errno.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <sys/time.h>
#include <sys/mman.h>
#include <sys/ioctl.h>
#include <linux/videodev2.h>

using namespace std;
using namespace cv;

int main(){
	int fd;

	//read and write access required and requested
	if((fd = open("/dev/video1",O_RDWR))<0){
		cout << "Error opening device." << endl;
		perror("open");
		exit(1);
	}

	struct v4l2_capability cap;
	if(ioctl(fd,VIDIOC_QUERYCAP, &cap) < 0){
		perror("VIDIOC_QUERYCAP");
		exit(1);
	}else{
		cout << "Capabilties of device retrieved." << endl;
	}

	//Checking video capabilities
	if(!(cap.capabilities & V4L2_CAP_VIDEO_CAPTURE)){
		cout << "The device does not handle single-planar video capture." << endl;
	}else{
		cout << "This device can handle video capture. Proceed." << endl;
	}

	if(!(cap.capabilities & V4L2_CAP_STREAMING)){
		cout << "This device does not allow video streaming." << endl;
	}else{
		cout << "This device allows video streaming. Proceed." << endl;
	}

	//Setting video format with IOCTL calls
	struct v4l2_format format;
	format.type = V4L2_BUF_TYPE_VIDEO_CAPTURE;
	format.fmt.pix.pixelformat = V4L2_PIX_FMT_MJPEG;
	format.fmt.pix.width = 1920;
	format.fmt.pix.height = 1080;

	if(ioctl(fd,VIDIOC_S_FMT, &format) < 0){
		perror("VIDIOC_S_FMT");
		exit(1);
	}else{
		cout << "Video format of device set. Proceed." << endl;
	}

	//Initializing buffer
	struct v4l2_requestbuffers buffrequest;
	buffrequest.type = V4L2_BUF_TYPE_VIDEO_CAPTURE;
	buffrequest.memory = V4L2_MEMORY_MMAP;
	buffrequest.count = 1;

	if(ioctl(fd,VIDIOC_REQBUFS, &buffrequest) < 0){
		perror("VIDIOC_REQBUFS");
		exit(1);
	}else{
		cout << "Buffer initialization complete. Proceed." << endl;
	}

	//Allocating buffers
	struct v4l2_buffer bufferinfo;
	memset(&bufferinfo, 0, sizeof(bufferinfo));

	bufferinfo.type = V4L2_BUF_TYPE_VIDEO_CAPTURE;
	bufferinfo.memory = V4L2_MEMORY_MMAP;
	bufferinfo.index = 0;

	if(ioctl(fd, VIDIOC_QUERYBUF, &bufferinfo) < 0){
		perror("VIDIOC_QUERYBUF");
		exit(1);
	}else{
		cout << "Buffer memory allocated. Proceed." << endl;
	}

	void *buffer_start = mmap(
			NULL,
			bufferinfo.length,
			PROT_READ | PROT_WRITE,
			MAP_SHARED,
			fd,
			bufferinfo.m.offset
			);

	if(buffer_start == MAP_FAILED){
		perror("mmap");
		exit(1);
	}

	memset(buffer_start, 0 ,bufferinfo.length);

	struct v4l2_buffer bufferinfo2;
	memset(&bufferinfo2, 0, sizeof(bufferinfo2));

	bufferinfo2.type = V4L2_BUF_TYPE_VIDEO_CAPTURE;
	bufferinfo2.memory = V4L2_MEMORY_MMAP;
	bufferinfo2.index = 0;

	int type = bufferinfo2.type;
	if(ioctl(fd, VIDIOC_STREAMON, &type) < 0){
		perror("VIDIOC_STREAMON");
		exit(1);
	}

	int key;
	namedWindow("image1",CV_WINDOW_AUTOSIZE);

	while(1){
		if(ioctl(fd, VIDIOC_QBUF, &bufferinfo2) < 0){
			perror("VIDIOC_QBUF");
			exit(1);
		}

		if(ioctl(fd ,VIDIOC_DQBUF, &bufferinfo2) < 0){
			perror("VIDIOC_DQBUF");
			exit(1);
		}

		Mat img(Size(1920, 1080), CV_8UC1, (uchar *)buffer_start);
		Mat image = imdecode(img,1);

		if(!image.empty()){
			imshow("image1",image);
			//imwrite("image.jpg",image);
		}else{
			cout << "Image is empty" << endl;
		}
		key=waitKey(10);
		if(key == 27){
			break;
		}
	}

	if(ioctl(fd, VIDIOC_STREAMOFF, &type) < 0){
		perror("VIDIOC_STREAMOFF");
		exit(1);
	}

	//Convert image in buffer memory to opencv Mat
	Mat img(Size(1280, 720), CV_8UC1, (uchar *)buffer_start);
	Mat image = imdecode(img,1);
	namedWindow("image1",CV_WINDOW_AUTOSIZE);

	if(!image.empty()){
		imshow("image1",image);
		waitKey(0);
		imwrite("image.jpg",image);
	}else{
		cout << "Image is empty" << endl;
	}

	cout << "Successful" << endl;

	close(fd);
	return EXIT_SUCCESS;
}*/


int main(){

	clock_t start;
	clock_t end;
	Capture cap1("/dev/video1");
	Capture cap2("/dev/video2");
	cap1.setWidth(1920);
	cap1.setHeight(1080);
	cap2.setWidth(1920);
	cap2.setHeight(1080);

	cap1.startCapture();
	cap2.startCapture();

	Mat image1;
	Mat image2;
	int key;
	namedWindow("image1",1);
	//namedWindow("image2",1);

	while(1){
		start = clock();
		image1 = cap1.grabFrame();
	//	image2 = cap2.grabFrame();
		end = clock();

		cout << "FPS:" << 1/((end-start)/(double)CLOCKS_PER_SEC) << endl;
		imshow("image1",image1);
		//imshow("image2",image2);

		key = waitKey(1);
		if(key == 27){
			break;
		}
	}
	return EXIT_SUCCESS;
}
