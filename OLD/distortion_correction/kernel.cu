#include "hip/hip_runtime.h"
 
#include <opencv2/core/cuda_devptrs.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/gpu/gpu.hpp>
#include <opencv2/gpu/stream_accessor.hpp>
#include <opencv2/opencv.hpp>
#include <opencv2/highgui/highgui.hpp>
#define RADIUS 340
#define centreX 320
#define centreY 240
#define PI 3.1415926
#define cameraFieldAngle PI
#define LIMIT 1e-4
using namespace cv;
using namespace cv::gpu;
using namespace std;
//�Զ����ں˺���
//void swap_rb_caller(const PtrStepSz<uchar3>& src,PtrStep<uchar3> dst,hipStream_t stream);
__device__ double func(double l, double phi)
{
	double result = (l / 2 - 1)*cos(phi) - sin(phi) + 1;
	return result;
}
__device__ double getPhi(double l)
{
	double head = 0;
	double tail = 0;
	double mid = 0;
	double result = 0;
	if (l >= 0 && l < 2)
	{
		head = 0;
		tail = PI / 2;
		mid = head;
		result = func(l, mid);
		while (abs(result)>LIMIT)
		{
			mid = (tail + head) / 2;
			result = func(l, mid);

			if (result > 0)
			{
				head = mid;
			}
			else
			{
				tail = mid;
			}
		}
	}
	else
	{
		head = PI / 2;
		tail = PI;
		mid = tail;
		result = func(l, mid);
		while (abs(result) > LIMIT)
		{
			mid = (tail + head) / 2;
			result = func(l, mid);
			if (result < 0)
			{
				head = mid;
			}
			else
			{
				tail = mid;
			}
		}
	}
	return mid;
}
__global__ void correction_kernel2(const PtrStepSz<uchar3> src, PtrStep<uchar3> dst)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	/*double dx = cameraFieldAngle / src.cols;
	double dy = cameraFieldAngle / src.rows;

	double latitude = getPhi((double)j * 4.0 / src.rows);
	double longitude = getPhi((double)i * 4.0 / src.cols);*/
	double dx = cameraFieldAngle / src.cols;
	double dy = dx;
	double latitude = (PI - cameraFieldAngle) / 2 + j * dy;
	double longitude = (PI - cameraFieldAngle) / 2 + i * dx;
	double x = -sin(latitude)*cos(longitude);
	double y = cos(latitude);
	double z = sin(latitude) * sin(longitude);
	double theta_shpere = acos(z);
	double phi_shpere = atan2(y, x);
	double foval = RADIUS / (cameraFieldAngle / 2);
	double p = foval * theta_shpere;
	double theta = phi_shpere;
	double x_cart = p * cos(theta);
	double y_cart = p * sin(theta);
	int u = x_cart + centreX;
	int v = -y_cart + centreY;
	if (i < src.cols && j < src.rows && u < src.cols && v < src.cols)
	{
		uchar3 value = src(v, u);
		dst(j, i) = make_uchar3(value.x, value.y, value.z);
	}
}
__global__ void correction_kernel(const PtrStepSz<uchar3> src, PtrStep<uchar3> dst)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	double dx = cameraFieldAngle / src.cols;
	double dy = dx;
	double latitude = (PI - cameraFieldAngle) / 2 + j * dy;
	double longitude = (PI - cameraFieldAngle) / 2 + i * dx;
	double x = -sin(latitude) * cos(longitude);
	double y = cos(latitude);
	double z = sin(latitude) * sin(longitude);

	double theta_shpere = acos(z);
	double phi_shpere = atan2(y, x);
	double p = sin(theta_shpere);
	double theta = phi_shpere;
	double x_cart = p * cos(theta);
	double y_cart = p * sin(theta);
	int u = x_cart * RADIUS + centreX;
	int v = -y_cart * RADIUS + centreY;
	if (i < src.cols && j < src.rows && u < src.cols && v < src.rows)
	{
		uchar3 value = src(v, u);
		dst(j, i) = make_uchar3(value.x, value.y, value.z);
	}
}

void correction_caller(const PtrStepSz<uchar3>& src, PtrStep<uchar3> dst, hipStream_t stream)
{
	/*dim3 block(32,8);

	dim3 grid((src.cols + block.x - 1) / block.x, (src.rows + block.y - 1) / block.y);

	correction_kernel <<<grid, block, 0, stream >>>(src, dst);
	if (stream == 0)
		hipDeviceSynchronize();
		*/
	int blockWidth = 16;
	int blockHeight = 8;
	dim3 blockSize(blockWidth, blockHeight);
	int   blocksY = src.rows / blockHeight + 1;
	int   blocksX = src.cols / blockWidth + 1; //TODO
	const dim3 gridSize(blocksX, blocksY);  //TODO
	correction_kernel2 <<<gridSize, blockSize, 0, stream >>>(src, dst);
	if (stream == 0)
	{
		hipDeviceSynchronize();
	}
}
void correction(const GpuMat& src, GpuMat& dst, Stream& stream = Stream::Null())
{
	CV_Assert(src.type() == CV_8UC3);
	Size imgSize(src.size().width, src.size().width);
	dst.create(imgSize, src.type());
	hipStream_t s = StreamAccessor::getStream(stream);
	correction_caller(src, dst, s);

}
int main() {
	Mat image0;
	image0 = imread("D:\\Documents\\Images\\capture.jpg");
	if (image0.empty()) {
		return -1;
	}
	Size imgSize(image0.size().width, image0.size().width);
	Mat dst(imgSize, CV_8UC3, Scalar(0, 0, 0));
	GpuMat gpuMat, output;
	gpuMat.upload(image0);//upload image to gpuMat
	correction(gpuMat, output);
	output.download(dst);
	imwrite("D:\\Documents\\Images\\corrected_image0.jpg", dst);
	printf("done\n");
	waitKey(0);
/*	VideoCapture cap0(1);
	if (!cap0.isOpened())
	{
		cout << "Cannot open the video cam 0" << endl;
		return -1;
	}

	while (1) {
		Mat image0;
		cap0.read(image0);
		if (!image0.empty())
		{
			imshow("image0", image0);
		}
		Size imgSize(image0.size().width, image0.size().width);
		Mat dst(imgSize, CV_8UC3, Scalar(0, 0, 0));
		//imshow("src",image);
		//if(!gpuMat.empty()){
		//free(gpuMat);
	//	}
		//if(!output.empty()){
		//free(output);
	//	}
		GpuMat gpuMat, output;
		gpuMat.upload(image0);//upload image to gpuMat
		correction(gpuMat, output);
		output.download(dst);
		imshow("corrected_image0", dst);
		if (waitKey(100) == 27) //wait for 'esc' key press for 30 ms. If 'esc' key is pressed, break loop
		{
			cout << "esc key is pressed by user" << endl;
			break;
		}
	}*/
	/*Mat image = imread("capture.jpg");
	Size imgSize(image.size().width,image.size().width);
	Mat dst(imgSize,CV_8UC3,Scalar(0,0,0));
	imshow("src",image);
	GpuMat gpuMat,output;
	gpuMat.upload(image);//upload image to gpuMat
	swap_rb(gpuMat,output);
	output.download(dst);*/
	//imwrite("result.jpg",dst);
	//waitKey(0);
	return 0;
}