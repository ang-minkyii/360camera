#include "hip/hip_runtime.h"
//swap_rb.cu
#include <opencv2/core/cuda_devptrs.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/gpu/gpu.hpp>
#include <opencv2/gpu/stream_accessor.hpp>
#include <opencv2/opencv.hpp>
#include <opencv2/highgui/highgui.hpp>
#define RADIUS 340
#define centreX 320
#define centreY 240
#define PI 3.1415926
#define cameraFieldAngle PI

using namespace cv;
using namespace cv::gpu;
using namespace std;
//自定义内核函数
//void swap_rb_caller(const PtrStepSz<uchar3>& src,PtrStep<uchar3> dst,hipStream_t stream);


__global__ void correction_kernel(const PtrStepSz<uchar3> src,PtrStep<uchar3> dst)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    double dx = cameraFieldAngle/src.cols;
    double dy = dx;
    double latitude = (PI - cameraFieldAngle)/2  + j * dy;
    double longitude = (PI - cameraFieldAngle)/2 + i * dx;
    double x = -sin(latitude) * cos(longitude);
    double y = cos(latitude);
    double z = sin(latitude) * sin(longitude);

    double theta_shpere = acos(z);
    double phi_shpere = atan2(y,x);
    double p = sin(theta_shpere);
    double theta = phi_shpere;
    double x_cart = p * cos(theta);
    double y_cart = p * sin(theta);
    int u = x_cart * RADIUS + centreX;
    int v = -y_cart * RADIUS + centreY;
    if(i < src.cols && j < src.rows && u < src.cols && v < src.rows)
    {
        uchar3 value = src(v,u);
        dst(j,i) = make_uchar3(value.x,value.y,value.z);
    }
}

void correction_caller(const PtrStepSz<uchar3>& src,PtrStep<uchar3> dst,hipStream_t stream)
{
    dim3 block(32,8);
    dim3 grid((src.cols + block.x - 1)/block.x,(src.rows + block.y - 1)/block.y);

    correction_kernel<<<grid,block,0,stream>>>(src,dst);
    if(stream == 0)
        hipDeviceSynchronize();
}
void correction(const GpuMat& src,GpuMat& dst,Stream& stream = Stream::Null())
{
    CV_Assert(src.type() == CV_8UC3);
    Size imgSize(src.size().width,src.size().width);
    dst.create(imgSize,src.type());
    hipStream_t s = StreamAccessor::getStream(stream);
    correction_caller(src,dst,s);

}
int main(){
	VideoCapture cap0(1);
	if (!cap0.isOpened())
	{
				cout << "Cannot open the video cam 0" << endl;
				return -1;
	}

	while(1){
		Mat image0;
		cap0.read(image0);
		if(!image0.empty())
		{
			imshow("image0",image0);
		}
		Size imgSize(image0.size().width,image0.size().width);
		Mat dst(imgSize,CV_8UC3,Scalar(0,0,0));
		//imshow("src",image);
		/*if(!gpuMat.empty()){
			free(gpuMat);
		}
		if(!output.empty()){
			free(output);
		}*/
		GpuMat gpuMat,output;
		gpuMat.upload(image0);//upload image to gpuMat
		correction(gpuMat,output);
		output.download(dst);
		imshow("corrected_image0",dst);
		if(waitKey(100) == 27) //wait for 'esc' key press for 30 ms. If 'esc' key is pressed, break loop
		{
			  cout << "esc key is pressed by user" << endl;
			  break;
		}
	}
	/*Mat image = imread("capture.jpg");
	Size imgSize(image.size().width,image.size().width);
	Mat dst(imgSize,CV_8UC3,Scalar(0,0,0));
	imshow("src",image);
	GpuMat gpuMat,output;
	gpuMat.upload(image);//upload image to gpuMat
	swap_rb(gpuMat,output);
	output.download(dst);*/
	//imwrite("result.jpg",dst);
	//waitKey(0);
	return 0;
}
